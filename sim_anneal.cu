#include "hip/hip_runtime.h"
// @file:     sim_anneal.cu
// @author:   Samuel
// @created:  2017.08.23
// @editted:  2018.06.13 - Robert
// @license:  GNU LGPL v3
//
// @desc:     Simulated annealing physics engine

#include "sim_anneal.h"
#include <ctime>

#include <boost/numeric/ublas/vector.hpp>
#include <boost/numeric/ublas/io.hpp>


//#define STEADY_THREASHOLD 700       //arbitrary value used in restarting

using namespace phys;

std::mutex siqadMutex;

__device__ hipblasHandle_t cb_hdl;
__device__ float mu, kT0, kT_step, v_freeze_step;

// CUDA error checking
#define cudaCheckErrors(msg) \
  if (hipGetLastError() != hipSuccess) { \
    std::cerr << "CUDA error: " << msg << "(" << hipGetErrorString(hipGetLastError()) \
      << " at " << __FILE__ << ":" << __LINE__ << ")" << std::endl; \
    exit(1); \
  }

// cuBLAS error checking
#define cublasCheckErrors(cublas_status) \
  if (cublas_status != HIPBLAS_STATUS_SUCCESS) { \
    std::cerr << "Fatal cuBLAS error: " << (int)(cublas_status) << "(at " << \
      __FILE__ << ":" << __LINE__ << ")" << std::endl; \
    exit(1); \
  }

// print 1d array content in device
#define print1DArray(name, arr, size) \
  printf("%s=[", name); \
  for (int i=0; i<size; i++) { \
    printf("%f", arr[i]); \
    if (i!=size-1) \
      printf(", "); \
  } \
  printf("]\n");


__global__ void simAnnealAlg(int n_dbs, float *v_ext, float *v_ij, float *mu, int t_max, float kT_init)
{
  // cycle tracking
  int t=0;                      // current anneal cycle

  // population related vars
  bool pop_changed;             // indicate whether population has changed during this cycle
  float v_freeze=0;             // current freeze out voltage (population)
  float kT=kT_init;             // current temperature (population)
  float *randnums;              // random numbers for probability evaluation
  randnums = (float*)malloc(n_dbs*sizeof(float));

  // hop related vars
  int from_occ_ind, to_occ_ind; // hopping from n[occ[from_ind]]
  int from_ind, to_ind;         // hopping from n[from_ind] to n[to_ind]
  int hop_attempts;

  float *n;                     // current occupation of DB sites
  float *dn;                    // change of occupation for population update
  float *v_local;               // local potential at each site
  n = (float*)malloc(n_dbs*sizeof(float));
  dn = (float*)malloc(n_dbs*sizeof(float));
  v_local = (float*)malloc(n_dbs*sizeof(float));

  for (int i=0; i<n_dbs; i++) {
    n[i] = 0.;
    dn[i] = 0.;
    v_local[i] = 0.;
    randnums[i] = 0.5;
  }

  // initialize system energy and local energy
  printf("Initializing system energy and local energy\n");
  float E_sys;
  systemEnergy(n_dbs, n, v_ext, v_ij, &E_sys);
  initVLocal(n_dbs, n, v_ext, v_ij, v_local);

  printf("\n***Beginning simanneal***\n");
  while (t < t_max) {
    // Population
    // TODO generate randnums
    //printf("Generating population delta\n");
    pop_changed = false;
    genPopulationDelta(n_dbs, n, v_local, &v_freeze, &kT, mu, randnums, dn, &pop_changed);
    __syncthreads();
    if (pop_changed) {

      // print dn
      print1DArray("Population changed, dn", dn, n_dbs);

      float alpha=1;
      // n + dn
      hipblasSaxpy(cb_hdl, n_dbs, &alpha, dn, 1, n, 1);
      __syncthreads();
      // E_sys += energy delta from population change
      populationChangeEnergyDelta(n_dbs, dn, v_ij, v_local, &E_sys);
      __syncthreads();
      // v_local = - prod(v_ij, dn) + v_local 
      alpha=-1;
      float beta=1;
      hipblasSgemv(cb_hdl, HIPBLAS_OP_N, n_dbs, n_dbs, &alpha, v_ij, n_dbs, dn, 1,
          &beta, v_local, 1);
      __syncthreads();

      // print new v_local
      print1DArray("v_local", v_local, n_dbs);

      // print new n
      print1DArray("n", n, n_dbs);
    }

    // TODO occupation list update

    // TODO Hopping

    // TODO store new arrangement

    // TODO time step
    timeStep(&t, &kT, &v_freeze);
    __syncthreads();
    printf("\n");
  }

  free(randnums);
  free(n);
  free(dn);
  free(v_local);
}


__global__ void initCublasHandle()
{
  hipblasCreate(&cb_hdl);
}

__global__ void destroyCublasHandle()
{
  hipblasDestroy(cb_hdl);
}

__global__ void initSimAnnealConsts(float *mu_in, float kT0_in, 
    float kT_step_in, float v_freeze_step_in)
{
  mu = *mu_in;
  kT0 = kT0_in;
  kT_step = kT_step_in;
  v_freeze_step = v_freeze_step_in;
  printf("setting mu = %f", mu);
  // TODO time step constants
}

__device__ void initVLocal(int n_dbs, float *n, float *v_ext, float *v_ij, float *v_local)
{
  // v_local = v_ext - dot(v_ij, n)

  // dot(v_ij, n)
  float alpha=1;
  float beta=0;
  hipblasSgemv(cb_hdl, HIPBLAS_OP_N, n_dbs, n_dbs, &alpha, v_ij, n_dbs, n, 1, 
      &beta, v_local, 1);

  // v_ext - above
  alpha=-1;
  hipblasSaxpy(cb_hdl, n_dbs, &alpha, v_local, 1, v_ext, 1);
}

__device__ void genPopulationDelta(int n_dbs, float *n, float *v_local, 
    float *v_freeze, float *kT, float *mu, float *randnum, float *dn, 
    bool *pop_changed)
{
  printf("Generating population delta. v_freeze=%f, kT=%f, mu=%f\n", *v_freeze, *kT, *mu);
  printf("prob=[");
  for (int i=0; i<n_dbs; i++) {
    // TODO consider replacing expf with __expf for faster perf
    float prob = 1. / ( 1. + expf( ((2.*n[i]-1.)*(v_local[i]+*mu) + *v_freeze ) / *kT ));
    if (randnum[i] < prob) {
      dn[i] = 1. - 2.*n[i];
      *pop_changed = true;
    } else {
      dn[i] = 0.;
    }
    printf("%f", prob);
    if (i != n_dbs-1)
      printf(", ");
  }
  printf("]\n");
}

// Total system energy including Coulombic repulsion and external voltage.
// NOTE keep this version around for benchmarking, this seems to be faster than cublas for some reason...
/*__global__ void systemEnergy(float *v, int n_dbs, float *n, float *v_ext, float *v_ij)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i=index; i<n_dbs; i+=stride) {
    *v -= v_ext[i] * n[i];
    for (int j=i+1; j<n_dbs; j++)
      *v += v_ij[i*n_dbs + j] * n[i] * n[j];
  }
}*/

__device__ void systemEnergy(int n_dbs, float *n, float *v_ext, float *v_ij, float *output)
{
  // TODO might be able to merge this function with population change energy delta with the similarity
  float *coulomb_v = (float*)malloc(sizeof(float));
  totalCoulombPotential(n_dbs, n, v_ij, coulomb_v);
  __syncthreads();
  hipblasSdot(cb_hdl, n_dbs, v_ext, 1, n, 1, output);
  __syncthreads();
  *output *= -1;
  *output += *coulomb_v;
  free(coulomb_v);
}

// Energy change from population change.
__device__ void populationChangeEnergyDelta(int n_dbs, float *dn, float *v_ij, float *v_local, float *output)
{
  // delta E = -1 * dot(v_local, dn) + dn^T * V_ij * dn
  float *coulomb_v = (float*)malloc(sizeof(float));
  totalCoulombPotential(n_dbs, dn, v_ij, coulomb_v);
  __syncthreads();
  hipblasSdot(cb_hdl, n_dbs, v_local, 1, dn, 1, output);
  __syncthreads();
  *output *= -1;
  *output += *coulomb_v;
  free(coulomb_v);
}

// Total potential from Coulombic repulsion in the system.
__device__ void totalCoulombPotential(int n_dbs, float *n, float *v_ij, float *v)
{
  float alpha=0.5;
  float beta=0;
  float *v_temp = (float*)malloc(n_dbs);
  hipblasStatus_t status = hipblasSgemv(cb_hdl, HIPBLAS_OP_N, n_dbs, n_dbs, &alpha, v_ij, n_dbs, n, 1, &beta, v_temp, 1);
  __syncthreads();
  status = hipblasSdot(cb_hdl, n_dbs, n, 1, v_temp, 1, v);
  __syncthreads();
  free(v_temp);
}

__device__ void hopEnergyDelta(int i, int j, int n_dbs, float *v_local, float *v_ij, float *v_del)
{
  *v_del = v_local[i] - v_local[j] - v_ij[IDX2C(i,j,n_dbs)];
}

__device__ void timeStep(int *t, float *kT, float *v_freeze)
{
  *t += 1;
  *kT = kT0 + (*kT - kT0) * kT_step;
  *v_freeze = (float)(*t) * v_freeze_step;
}



//Global method for writing to vectors (global in order to avoid thread clashing).
void writeStore(SimAnneal *object, int threadId){
  siqadMutex.lock();

  object->chargeStore[threadId] = object->db_charges;
  object->energyStore[threadId] = object->config_energies; object->numElecStore[threadId] = object->n_elec;

  siqadMutex.unlock();
}

SimAnneal::SimAnneal(const int thread_id)
{
  rng.seed(std::time(NULL)*thread_id+4065);
  dis01 = boost::random::uniform_real_distribution<float>(0,1);
  threadId = thread_id;
}

void SimAnneal::runSim()
{
  // initialize variables & perform pre-calculation
  kT = 300*constants::Kb;    // kT = Boltzmann constant (eV/K) * 298 K
  v_freeze = 0;

  // resize vectors
  v_local.resize(n_dbs);

  db_charges.resize(result_queue_size);
  n.resize(n_dbs);
  occ.resize(n_dbs);

  config_energies.resize(result_queue_size);

  // SIM ANNEAL
  simAnneal();
}

void SimAnneal::runSimCUDA()
{
  // initialize variables & perform pre-calculations
  float kT = 300*constants::Kb; // kT = Boltzmann constant (eV/K) * 300K
  
  float *d_v_ext, *d_v_ij, *d_mu;
  hipMallocManaged(&d_v_ext, n_dbs*sizeof(float));
  hipMallocManaged(&d_v_ij, n_dbs*n_dbs*sizeof(float));
  hipMallocManaged(&d_mu, sizeof(float));

  for (int i=0; i<n_dbs; i++) {
    d_v_ext[i] = v_ext[i];
    for (int j=0; j<n_dbs; j++) {
      d_v_ij[IDX2C(i,j,n_dbs)] = v_ij(i,j);
    }
  }

  std::cout << "initializing cublas handle" << std::endl;
  ::initCublasHandle<<<1,1>>>();
  hipDeviceSynchronize();

  std::cout << "initializing CUDA SimAnneal constants" << std::endl;
  *d_mu = mu;
  ::initSimAnnealConsts<<<1,1>>>(d_mu, kT0, kT_step, v_freeze_step);
  hipDeviceSynchronize();

  std::cout << "invoking CUDA SimAnneal..." << std::endl;
  //::simAnnealAlg<<<1,1>>>(n_dbs, d_v_ext, d_v_ij, t_max, kT);
  ::simAnnealAlg<<<1,1>>>(n_dbs, d_v_ext, d_v_ij, d_mu, t_max, kT);
  hipDeviceSynchronize();

  std::cout << "destroying cublas handle" << std::endl;
  ::destroyCublasHandle<<<1,1>>>();
  hipDeviceSynchronize();

  // TODO move results to a form understood by SiQADConn
  
  // clean up
  hipFree(d_v_ext);
  hipFree(d_v_ij);
  hipFree(d_mu);
}


void SimAnneal::simAnneal()
{
  // Vars
  boost::numeric::ublas::vector<int> dn(n_dbs); // change of occupation for population update
  int from_occ_ind, to_occ_ind; // hopping from n[occ[from_ind]]
  int from_ind, to_ind;         // hopping from n[from_ind] to n[to_ind]
  int hop_attempts;

  //n_best.resize(n.size());      //Variables used in restarting. uncomment
  //firstBest = false;            //these two for restarts.

  E_sys = systemEnergy();
  //E_best = E_sys;         // initializing the best system energy with the initial energy
  //n_best = n;             //initializing the best electron configuration with the initial electron config.
  v_local = v_ext - ublas::prod(v_ij, n);

  //steadyPopCount = 0;           //Variable for restarting. Uncomment when restarting.

  
  /*
  // arrays for CUDA code
  float *n_arr, *v_ext_arr, *v_ij_arr, *cuda_v, *dn_arr, *v_local_arr;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&n_arr, n_dbs*sizeof(float));
  hipMallocManaged(&v_local_arr, n_dbs*sizeof(float));
  hipMallocManaged(&v_ext_arr, n_dbs*sizeof(float));
  hipMallocManaged(&v_ij_arr, n_dbs*n_dbs*sizeof(float));
  hipMallocManaged(&cuda_v, sizeof(float));
  hipMallocManaged(&dn_arr, n_dbs*sizeof(float));

  ::initCublasHandle<<<1, 1>>>();
  */


  // Run simulated annealing for predetermined time steps
  while(t < t_max) {

    // Population
    dn = genPopDelta();

    bool pop_changed = false;
    for (unsigned i=0; i<dn.size(); i++) {
      if (dn[i] != 0) {
        pop_changed = true;
        break;
      }
    }

/*        //Used in Restarting. Uncomment if restarting.
    if(pop_changed){
      steadyPopCount = 0;
    }
    else{
      steadyPopCount++;
    }
*/

    if (pop_changed) {
      n += dn;
      E_sys += -1 * ublas::inner_prod(v_local, dn) + totalCoulombPotential(dn);
      v_local -= ublas::prod(v_ij, dn);



      /*
      // NOTE CUDA test
      float c_ver = totalCoulombPotential(dn);
      float c_E_sys_del = -1 * ublas::inner_prod(v_local, dn) + totalCoulombPotential(dn);
      //float c_E_sys_del = -1 * ublas::inner_prod(v_local, dn);

      // copy vector data to array
      for (int i=0; i<n_dbs; i++) {
        dn_arr[i] = dn[i];
        n_arr[i] = n[i];
        v_ext_arr[i] = v_ext[i];
        v_local_arr[i] = v_local[i];
        for (int j=0; j<n_dbs; j++) {
          v_ij_arr[IDX2C(i,j,n_dbs)] = v_ij(i,j);
        }
      }
      *cuda_v = 0;
      ::totalCoulombPotential<<<1, 1>>>(n_dbs, dn_arr, v_ij_arr, cuda_v);
      hipDeviceSynchronize();

      std::cout << "total coulomb potential c++ :   " << c_ver << std::endl;
      std::cout << "total coulomb potential cuda:   " << *cuda_v << std::endl;
      std::cout << std::endl;

      *cuda_v = 0;
      ::populationChangeEnergyDelta<<<1, 1>>>(n_dbs, dn_arr, v_ij_arr, v_local_arr, cuda_v);
      hipDeviceSynchronize();

      std::cout << "pop change c++ : " << c_E_sys_del << std::endl;
      std::cout << "pop change cuda: " << *cuda_v << std::endl;
      std::cout << std::endl;
      */
    }


    // Occupation list update
    int occ_ind=0, unocc_ind=n_dbs-1;
    for (int db_ind=0; db_ind<n_dbs; db_ind++) {
      if (n[db_ind])
        occ[occ_ind++] = db_ind;
      else
        occ[unocc_ind--] = db_ind;
    }
    n_elec = occ_ind;


    // Hopping
    hop_attempts = 0;
    if (n_elec != 0) {
      while (hop_attempts < (n_dbs-n_elec)*5) {
        from_occ_ind = getRandOccInd(1);
        to_occ_ind = getRandOccInd(0);
        from_ind = occ[from_occ_ind];
        to_ind = occ[to_occ_ind];

        float E_del = hopEnergyDelta(from_ind, to_ind);
        if (acceptHop(E_del)) {
          performHop(from_ind, to_ind);
          occ[from_occ_ind] = to_ind;
          occ[to_occ_ind] = from_ind;
          // calculate energy difference
          E_sys += E_del;
          ublas::matrix_column<ublas::matrix<float>> v_i (v_ij, from_ind);
          ublas::matrix_column<ublas::matrix<float>> v_j (v_ij, to_ind);
          v_local += v_i - v_j;
        }
        hop_attempts++;
      }
    }

    // push back the new arrangement
    db_charges.push_back(n);
    config_energies.push_back(E_sys);

    // perform time-step if not pre-annealing
    timeStep();
  }

  /*
  // copy vector data to array
  for (int i=0; i<n_dbs; i++) {
    n_arr[i] = n[i];
    v_ext_arr[i] = v_ext[i];
    std::cout << "n_arr[" << i << "] = " << n_arr[i] << std::endl;
    std::cout << "v_ext_arr[" << i << "] = " << v_ext_arr[i] << std::endl;
    for (int j=0; j<n_dbs; j++) {
      v_ij_arr[i*n_dbs + j] = v_ij(i,j);
      std::cout << "v_ij_arr[" << i*n_dbs+j << "] = " << v_ij_arr[i*n_dbs+j] << std::endl;
    }
  }

  //*cuda_v = 0;
  //::systemEnergy<<<1, 1>>>(cuda_v, n_dbs, n_arr, v_ext_arr, v_ij_arr);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  std::cout << "Host systemEnergy()=" << systemEnergy() << std::endl;
  //std::cout << "CUDA systemEnergy()=" << *cuda_v << std::endl;

  *cuda_v = 0;
  //::systemEnergyCublas<<<1, 1>>>(n_dbs, n_arr, v_ext_arr, v_ij_arr, cuda_v);
  ::systemEnergy<<<1, 1>>>(n_dbs, n_arr, v_ext_arr, v_ij_arr, cuda_v);
  hipDeviceSynchronize();
  std::cout << "cublas systemEnergy=" << *cuda_v << std::endl;
  
  // Free memory
  hipFree(n_arr);
  hipFree(v_ext_arr);
  hipFree(v_ij_arr);
  hipFree(cuda_v);
  hipFree(dn_arr);
  ::destroyCublasHandle<<<1, 1>>>();*/

  writeStore(this, threadId);
}










ublas::vector<int> SimAnneal::genPopDelta()
{
  ublas::vector<int> dn(n_dbs);
  for (unsigned i=0; i<n.size(); i++) {
    //float prob = 1. / ( 1 + exp( ((2*n[i]-1)*v_local[i] + v_freeze) / kT ) );
    float prob = 1. / ( 1 + exp( ((2*n[i]-1)*(v_local[i] + mu) + v_freeze) / kT ) );
    std::cout << "n[i]=" << n[i] << ", v_local[i]=" << v_local[i] 
      << ", mu=" << mu << ", v_freeze=" << v_freeze << ", kT=" << kT 
      << ", prob=" << prob << std::endl;
    dn[i] = evalProb(prob) ? 1 - 2*n[i] : 0;
  }
  return dn;
}

void SimAnneal::performHop(int from_ind, int to_ind)
{
  n[from_ind] = 0;
  n[to_ind] = 1;
}


void SimAnneal::timeStep()
{
  t++;
  kT = kT0 + (kT - kT0) * kT_step;
  v_freeze = t * v_freeze_step;

/*
  //simAnneal restarts
  if(!firstBest){
    firstBest = true;
    E_best = E_sys;
  }

  if(steadyPopCount > STEADY_THREASHOLD && E_sys < E_best){
    E_best = E_sys;
    n_best = n;
  }


  if( steadyPopCount > STEADY_THREASHOLD && (E_sys > 1.1*E_best || evalProb(0)) && t < 0.99*t_max){
    //t-=0.05*t_max;
    E_sys = E_best;
    n = n_best;
    std::cout << "******************RESTART******************" << std::endl;
  }
*/
}

// ACCEPTANCE FUNCTIONS

// acceptance function for hopping
bool SimAnneal::acceptHop(float v_diff)
{
  if (v_diff < 0)
    return true;

  // some acceptance function, acceptance probability falls off exponentially
  float prob = exp(-v_diff/kT);

  return evalProb(prob);
}


// takes a probability and generates true/false accordingly
bool SimAnneal::evalProb(float prob)
{
  //float generated_num = dis01(rng);
  boost::variate_generator<boost::random::mt19937&, boost::random::uniform_real_distribution<float>> rnd_gen(rng, dis01);

  return prob >= rnd_gen();
}





// ACCESSORS


int SimAnneal::getRandOccInd(int charge)
{
  int min,max;
  if (charge) {
    min = 0;
    max = n_elec-1;
  } else {
    min = n_elec;
    max = n_dbs-1;
  }
  boost::random::uniform_int_distribution<int> dis(min,max);
  return dis(rng);
}





// PHYS CALCULATION


float SimAnneal::systemEnergy()
{
  assert(n_dbs > 0);
  float v = 0;
  for(int i=0; i<n_dbs; i++) {
    //v -= mu + v_ext[i] * n[i];
    v -= v_ext[i] * n[i];
    for(int j=i+1; j<n_dbs; j++)
      v += v_ij(i,j) * n[i] * n[j];
  }
  return v;
}


float SimAnneal::distance(const float &x1, const float &y1, const float &x2, const float &y2)
{
  return sqrt(pow(x1-x2, 2.0) + pow(y1-y2, 2.0));
}


float SimAnneal::totalCoulombPotential(ublas::vector<int> config)
{
  return 0.5 * ublas::inner_prod(config, ublas::prod(v_ij, config));
}


float SimAnneal::interElecPotential(const float &r)
{
  //return exp(-r/debye_length) / r;
  return constants::Q0 * Kc * erf(r/constants::ERFDB) * exp(-r/debye_length) / r;
}


float SimAnneal::hopEnergyDelta(const int &i, const int &j)
{
  return v_local[i] - v_local[j] - v_ij(i,j);
}
